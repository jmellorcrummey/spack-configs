#include "hip/hip_runtime.h"
#include "gputest.h"
#include <hip_runtime.h>
#include <hip_runtime_api.h>
#include <hip/device_functions.h>

#if 0
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#endif

int checkxfers();

__global__ void
xcompute(const double *d_l1, const double *d_r1, double *d_p1, int nelements )
{
  dim3 blockDim;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < nelements) {
    d_p1[i] = d_p1[i] + 1.+ (sqrt( exp( log (d_l1[i]*d_l1[i]) ) + exp( log (d_r1[i]*d_r1[i]) ) ) ) /
        ( sqrt (exp( log(d_l1[i]*d_r1[i]) ) + exp( log( (d_r1[i]*d_l1[i]) )) ) );
  }
}

void
twork( int iter, int threadnum)
{
  double *l1 = lptr[threadnum];
  double *r1 = rptr[threadnum];
  double *p1 = pptr[threadnum];

  hipError_t err = hipSuccess;

  //allocate device memory for copying in l1 and r1, copying out p1
  size_t size = nn * sizeof(double);
  double *d_l1 = NULL;
  double *d_r1 = NULL;
  double *d_p1 = NULL;
  err = hipMalloc((void **)&d_l1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector d_l1 (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Allocated device vector d_l1\n" );
#endif
  }
  err = hipMalloc((void **)&d_r1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector d_r1 (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Allocated device vector d_r1\n" );
#endif
  }
  err = hipMalloc((void **)&d_p1, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector d_p1 (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Allocated device vector d_p1\n" );
#endif
  }

  // Copy l1,r1 and p1 to the device
  err = hipMemcpy(d_l1, l1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy device l1 to d_l1 (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Copied l1 to device vector d_l1\n" );
#endif
  }
  err = hipMemcpy(d_r1, r1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy device r1 to d_r1 (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Copied r1 to device vector d_r1\n" );
#endif
  }
  err = hipMemcpy(d_p1, p1, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy device p1 to d_p1 (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Copied p1 to device vector d_p1\n" );
#endif
  }

  // Set up and launch the CUDA kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = ( nn + threadsPerBlock -1 ) / threadsPerBlock;

  xcompute<<<blocksPerGrid, threadsPerBlock>>>(d_l1, d_r1, d_p1, nn);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch compute kernel (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Ran GPU kernel for xcompute\n" );
#endif
  }

  // Copy p1 back to the host
  err = hipMemcpy(p1, d_p1, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy p1 from device (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Copied d_p1 back to host\n" );
#endif
  }

  // Free the device memory
  err = hipFree(d_l1);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free d_l1 from device (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Freed device vector d_l1\n" );
#endif
  }
  err = hipFree(d_r1);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free d_r1 from device (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Freed device vector d_r1\n" );
#endif
  }
  err = hipFree(d_p1);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free d_p1 from device (error code %s)!\n", hipGetErrorString(err));
    exit(-1);
#if 0
  } else {
    fprintf(stderr, "Freed device vector d_p1\n" );
#endif
  }
#if 0
  fprintf(stderr, "\nCompleted iteration %d, thread %d\n\n", iter, threadnum);
#endif
}

void
checkgpu()
{
#if 0
  /* determine number of GPU's */
  int numdev = omp_get_num_devices();
  fprintf (stderr, "Machine has %d GPU device%s\n", numdev, (numdev==1 ? "" : "s") );

  /* Test if GPU is available */
  int	idev = omp_is_initial_device();

  int runningOnGPU = -1;
  #pragma omp target map(from:runningOnGPU)
  {
    runningOnGPU = omp_is_initial_device();
  }

  /* If still running on CPU, GPU must not be available */
  if (runningOnGPU != 0) {
#ifndef IGNORE_BAD_INITIAL_DEVICE
    fprintf(stderr, " ERROR unable to use the GPU! idev = %d, runningOnGpU -- omp_is_initial_device() = %d; exiting\n", idev, runningOnGPU);
    exit(1);
#else
    fprintf(stderr, " ERROR IGNORED idev = %d, runningOnGpU -- omp_is_initial_device() = %d; trying anyway\n", idev, runningOnGPU);
#endif
  } else {
    fprintfstderr, "    gputest is able to use the GPU! idev = %d, runningOnGpU -- omp_is_initial_device()\n", idev );
  }

  int ret = checkxfers();
  if (ret != 0 ) {
    fprintf(stderr, "Return from checkxfers = %d\n", ret);
  }
#endif
}

int
checkxfers()
{
// This is only needed for the OpenMP version
  return 0;
}
